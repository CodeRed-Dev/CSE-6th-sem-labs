#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

__global__ void reverseGrayLevel(unsigned char *img, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < numRows && col < numCols)
    {
        int index = row * numCols + col;
        img[index] = 255 - img[index];
    }
}

int main(int argc, char **argv)
{
    // Check for correct number of arguments
    if (argc != 3)
    {
        printf("Usage: %s input_image output_image\n", argv[0]);
        exit(1);
    }
    
    // Open input image file
    FILE *fp = fopen(argv[1], "rb");
    if (!fp)
    {
        printf("Could not open input image file %s\n", argv[1]);
        exit(1);
    }
    
    // Read in PGM header information
    char magic[3];
    int numCols, numRows, maxVal;
    fscanf(fp, "%2s %d %d %d", magic, &numCols, &numRows, &maxVal);
    
    // Allocate memory for input and output images
    int imageSize = numRows * numCols;
    unsigned char *h_inputImg = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
    unsigned char *h_outputImg = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
    unsigned char *d_inputImg, *d_outputImg;
    hipMalloc(&d_inputImg, imageSize * sizeof(unsigned char));
    hipMalloc(&d_outputImg, imageSize * sizeof(unsigned char));
    
    // Read in input image data
    fread(h_inputImg, sizeof(unsigned char), imageSize, fp);
    fclose(fp);
    
    // Copy input image data to device
    hipMemcpy(d_inputImg, h_inputImg, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    // Set up grid and block sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridSize((numCols + BLOCK_SIZE - 1) / BLOCK_SIZE, (numRows + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
    
    // Call kernel to reverse gray-level of image
    reverseGrayLevel<<<gridSize, blockSize>>>(d_inputImg, numRows, numCols);
    
    // Copy result back to host
    hipMemcpy(h_outputImg, d_inputImg, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    // Write out result to output image file
    fp = fopen(argv[2], "wb");
    fprintf(fp, "%s\n%d %d\n%d\n", magic, numCols, numRows, maxVal);
    fwrite(h_outputImg, sizeof(unsigned char), imageSize, fp);
    fclose(fp);
    
    // Free memory
    free(h_inputImg);
    free(h_outputImg);
    hipFree(d_inputImg);
    hipFree(d_outputImg);
    
    return 0;
}
//nvcc -o reverse_gray_level_image reverse_gray_level_image.cu
// ./reverse_gray_level_image input_image.pgm output_image.pgm
