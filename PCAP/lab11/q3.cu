#include<stdio.h>
#include "hip/hip_runtime.h"

#define TILE_WIDTH 2 
__global__ void MatrixMulKernel(int* d_M, int* d_N, int* d_P, int Width) {
 __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
 __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
 int bx = blockIdx.x; int by = blockIdx.y;
 int tx = threadIdx.x; int ty = threadIdx.y;
// Identify the row and column of the d_P element to work on
 int Row = by * TILE_WIDTH + ty;
 int Col = bx * TILE_WIDTH + tx;
 int Pvalue = 0;
// Loop over the d_M and d_N tiles required to compute d_P element
 for (int m = 0; m < Width/TILE_WIDTH; ++m) {
// Coolaborative loading of d_M and d_N tiles into shared memory
 Mds[ty][tx] = d_M[Row*Width + m*TILE_WIDTH + tx];
 Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty)*Width + Col];
 __syncthreads();
 for (int k = 0; k < TILE_WIDTH; ++k) {
 Pvalue += Mds[ty][k] * Nds[k][tx];
}
 __syncthreads();
}
 d_P[Row*Width + Col] = Pvalue;
}
void vecAdd(int* A, int *B, int* C,int m,int n,int p,int q)
{    int size1 = (m*n) * sizeof(int);
 int size2 = (p*q) * sizeof(int);
  int size3 = (m*q) * sizeof(int);
    //printf("elements m,n are %d%d%d",elements,m,n);
     int *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, size1);
    hipMemcpy(d_A, A, size1, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size2);
    hipMemcpy(d_B, B, size2, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_C, size3);
      // Kernel invocation code
 dim3 grid(ceil(n/((TILE_WIDTH)*1.0)),ceil(n/((TILE_WIDTH)*1.0)),1);
 dim3 blck(TILE_WIDTH,TILE_WIDTH,1);
    MatrixMulKernel<<<grid,blck>>> (d_A, d_B,d_C,n);
     hipMemcpy(C, d_C, size3,hipMemcpyDeviceToHost);
     // Free device memory for A, B, C
    hipFree(d_A); hipFree (d_C);
}
int main()
{
// Host data buffers
int *A = NULL;       // Input array
int *B = NULL;       // Input array
int *C = NULL;       // Output array
// Elements in each array
 int n=4,m=4;
 int p=4,q=4;
const int elements = m*n; // size of host data buffers
// Compute the size of the data in bytes
const int elements1 = p*q;
size_t datasize = sizeof(int)*elements;
size_t datasize1 = sizeof(int)*elements1;
// Dynamically allocate space for input/output host data buffers
A = (int*)malloc(datasize);
B = (int*)malloc(datasize1);
C = (int*)malloc(sizeof(int)*(m*q));
 int D[m][q];
// Initialize the input data
for(int i = 0; i < elements; i++)
{
   A[i] = i+1;
}
 for(int i = 0; i < elements1; i++)
{
   B[i] = i+1;
}
 for(int i = 0; i < elements; i++)
{ 
    printf("%d ",A[i]);
}
 for(int i = 0; i < elements1; i++)
{ 
    printf("%d ",B[i]);
}
 printf("\n");
vecAdd( A,B, C,m,n,p,q);
// Verify the output
bool result = true;
 int j,k=0;
for(int i = 0; i < m; i++)
{ 
    for(j=0;j<q;j++)
    D[i][j]=C[k++];
}
 for(int i = 0; i < m; i++)
{
    for(j=0;j<q;j++)
    printf("%d ",D[i][j]);
    printf("\n");
}
if(result)
        printf("Output is correct\n");
 else
     printf("Output is incorrect\n");
free(A); free(B); free(C);
return 0;
}