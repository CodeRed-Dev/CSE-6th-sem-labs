#include <stdio.h>
#include "hip/hip_runtime.h"

#define TW 4
#define MMW 5

__global__ void tiled1dconv(int *N, int *P, int *M, int MW, int width)
{
    __shared__ int Nds[TW + MMW - 1];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int hli, hri;
    int n = MW / 2;
    // loading left halo elements/ghost elements
    hli = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= (blockDim.x - n))
    {
        Nds[threadIdx.x - (blockDim.x - n)] = (hli < 0) ? 0 : N[hli];
    }
    // load center elements of the tiles
    Nds[n + threadIdx.x] = N[blockIdx.x * blockDim.x + threadIdx.x];
    // load right halo/ghost elements
    hri = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < n)
    {
        Nds[blockDim.x + n + threadIdx.x] = (hri > width) ? 0 : N[hri];
    }
    __syncthreads();
    int j, sum = 0;
    for (j = 0; j < MW; j++)
        sum = sum + M[j] * Nds[threadIdx.x + j];
    P[i] = sum;
}

int main()
{
    int A[16] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    int Mask[5] = {1, 0, 1, 0, 1};
    int width = 16;
    int mw = 5;
    int *P;
    int *d_A, *d_m, *d_P;
    int size1 = width * (sizeof(int));
    int size2 = mw * (sizeof(int));
    P = (int *)malloc(size1);
    hipMalloc((void **)&d_A, size1);
    hipMemcpy(d_A, A, size1, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_m, size2);
    hipMemcpy(d_m, Mask, size2, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_P, size1);
    dim3 blck(4, 1, 1);
    dim3 grid(4, 1, 1);
    tiled1dconv<<<grid, blck>>>(d_A, d_P, d_m, mw, width);
    hipMemcpy(P, d_P, size1, hipMemcpyDeviceToHost);
    for (int i = 0; i < width; i++)
        printf("%d  ", P[i]);
    printf("\n");
}