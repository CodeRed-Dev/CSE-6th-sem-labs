#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void word_count_kernel(char* str,char* key,int* word_indices,int* result)
{
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    int p=word_indices[idx];
    int n=word_indices[idx+1];
    char word[100];
    int i=0; 
    for(i=0;i<(n-p-1);i++)
    {
        word[i]=str[p+1+i]; //Extract the word from the input string
    }
    word[i]='\0'; //Terminator character for the word string
    int i1 = 0;
    int i2 = 0;
    int is_equal = 1;

    //Check if the word matches with the key word
    while (word[i1] != '\0' && key[i2] != '\0')
    {

        if (word[i1] == key[i2])
            {
                i1++;
                i2++;
            }else{
                is_equal = 0;
                break;
                }
    }

    if (is_equal == 1)
    {
        atomicAdd(result, 1); //increment result atomically if the word matches with the key word
    }
}

int main()
{
    char str[100]="I scream, you scream for ice cream"; //Input string
    char key[100]="scream"; //Key word
    //enter the string  
    //scanf("%s",str);
    int str_len=strlen(str); //Calculate the length of the input string
    int key_len=strlen(key); //Calculate the length of the key word
    int wc=0;
    for(int i=0;i<str_len;i++)
    {
        if(str[i]==' ')
        {
            wc++; //Count the number of spaces to determine the number of words in the input string
        }
    }
    wc--; //Exclude the last word
    int result=0; //Initialize the result to 0
    int* word_indices=(int*)(malloc(wc*sizeof(int))); //Allocate memory to store the indices of the words in the input string
    int wi=0;
    for(int i=0;i<str_len;i++)
    {
        if(str[i]==' '){
        word_indices[wi++]=i; //Store the index of the word in the input string
        }
    }
    
    char *d_str,*d_key;
    int *d_word_incdies,*d_result;
    hipMalloc((void**)&d_str,str_len*sizeof(char)); //Allocate memory for input string on the device
    hipMalloc((void**)&d_key,key_len*sizeof(char)); //Allocate memory for the key word on the device
    hipMalloc((void**)&d_word_incdies,(wc+1)*sizeof(int)); //Allocate memory for the word indices on the device
    hipMalloc((void**)&d_result,sizeof(int)); //Allocate memory for the result on the device
    hipMemcpy(d_str,str,str_len*sizeof(char),hipMemcpyHostToDevice); //Copy input string to device memory
    hipMemcpy(d_key,key,key_len*sizeof(char),hipMemcpyHostToDevice); //Copy key word to device memory
    hipMemcpy(d_word_incdies,word_indices,(wc+1)*sizeof(int),hipMemcpyHostToDevice); //Copy word indices to device memory
    hipMemcpy(d_result,&result,sizeof(int),hipMemcpyHostToDevice); //Copy result to device memory
    word_count_kernel<<<1,wc>>>(d_str,d_key,d_word_incdies,d_result); //Invoke the kernel with a single block and number of threads equal to
    hipMemcpy(&result,d_result,sizeof(int),hipMemcpyDeviceToHost);
    printf("%d ",result);
    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_result);
}
