#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void word_count_kernel(char* str,char* key,int* word_indices,int* result)
{
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    int p=word_indices[idx];
    int n=word_indices[idx+1];
    char word[100];
    int i=0; 
    for(i=0;i<(n-p-1);i++)
    {
        word[i]=str[p+1+i];
    }
    word[i]='\0';
    int i1 = 0;
    int i2 = 0;
    int is_equal = 1;

    while (word[i1] != '\0' && key[i2] != '\0')
    {

        if (word[i1] == key[i2])
            {
                i1++;
                i2++;
            }else{
                is_equal = 0;
                break;
                }
    }

    if (is_equal == 1)
    {
        atomicAdd(result, 1);
    }
}

int main()
{
    char str[100]="I scream, you scream for ice cream";
    char key[100]="scream";
    //enter the string  
    //scanf("%s",str);
    int str_len=strlen(str);
    int key_len=strlen(key);
    int wc=0;
    for(int i=0;i<str_len;i++)
    {
        if(str[i]==' ')
        {
            wc++;
        }
    }
    wc--;
    int result=0;
    int* word_indices=(int*)(malloc(wc*sizeof(int)));
    int wi=0;
    for(int i=0;i<str_len;i++)
    {
        if(str[i]==' '){
        word_indices[wi++]=i;
        }
    }
    
    char *d_str,*d_key;
    int *d_word_incdies,*d_result;
    hipMalloc((void**)&d_str,str_len*sizeof(char));
    hipMalloc((void**)&d_key,key_len*sizeof(char));
    hipMalloc((void**)&d_word_incdies,(wc+1)*sizeof(int));
    hipMalloc((void**)&d_result,sizeof(int));
    hipMemcpy(d_str,str,str_len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_key,key,key_len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_word_incdies,word_indices,(wc+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_result,&result,sizeof(int),hipMemcpyHostToDevice);
    word_count_kernel<<<1,wc>>>(d_str,d_key,d_word_incdies,d_result);
    hipMemcpy(&result,d_result,sizeof(int),hipMemcpyDeviceToHost);
    printf("%d ",result);
    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_result);
}