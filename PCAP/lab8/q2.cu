#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Kernel function to copy substrings of input string to output string
__global__ void kernel(char* sin, int* sin_len, char* sout)
{
    // Calculate the global index for the current thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int si = 0; // start index
    for (int i = 0; i < idx; i++)
    {
        // Calculate the start index for the substring that this thread will copy
        si += (*sin_len)-i;
    }

    // Calculate the total number of characters in the substring that this thread will copy
    int total_chars = (*sin_len) - idx;
    for (int i = 0; i < total_chars; i++)
    {
        // Copy each character of the substring from input to output string
        sout[si++] = sin[i];
    }
}

int main()
{
    char sin[100] = "PCAP"; // Input string
    char sout[100]; // Output string

    int sin_len = strlen(sin); // Length of input string
    int sout_len = 0;

    for (int i = 0; i < sin_len; i++)
    {
        sout_len += (i+1); // Calculate the total length of output string
    }

    // Allocate memory on device for input string, input string length, and output string
    char* d_sin;
    int* d_sin_len;
    char* d_sout;

    hipMalloc((void**) &d_sin, sin_len * sizeof(char));
    hipMalloc((void**) &d_sin_len, sizeof(int));
    hipMalloc((void**) &d_sout, (sout_len + 1) * sizeof(char)); // Add 1 to allocate space for null terminator

    // Copy input string, input string length, and output string to device
    hipMemcpy(d_sin, sin, sin_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_sin_len, &sin_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sout, sout, (sout_len + 1) * sizeof(char), hipMemcpyHostToDevice);

    // Call kernel function with 1 block and sin_len threads
    kernel<<<1, sin_len>>>(d_sin, d_sin_len, d_sout);

    // Copy output string from device to host
    hipMemcpy(sout, d_sout, (sout_len + 1) * sizeof(char), hipMemcpyDeviceToHost);

    // Add null terminator to output string
    sout[sout_len] = '\0';

    // Print input and output strings
    printf("String-input: %s\n", sin);
    printf("String-output: %s\n", sout);

    // Free device memory
    hipFree(d_sin);
    hipFree(d_sin_len);
    hipFree(d_sout);

    return 0;
}
