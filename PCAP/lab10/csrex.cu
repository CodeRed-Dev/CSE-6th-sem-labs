#include<stdio.h>
#include "hip/hip_runtime.h"


__global__ void kcsr(int *A,int *X,int *Y,int *colindex,int *rptr,int m)
{
    int row=threadIdx.x;
    int sum;
   for(int i=0;i<m+1;i++)
   printf("insdei kernel, %d ", rptr[i]);
     if(row<m)
     {
         sum=0;
         int rstart=rptr[row];
         int rend=rptr[row+1];
         for(int i=rstart;i<rend;i++)
         sum=sum+A[i]*X[colindex[i]];
      Y[row]=Y[row]+sum;
     }
}
int main()
{
    int A[4][4]={{3,0,1,0},{0,0,0,0},{0,2,4,1},{1,0,0,1}};
    int col_index[4],csr[10];
    
    int *X,*Y;
   int M=4,N=4;
 int vsize=M*sizeof(int);
 X=(int *)malloc(vsize);
 Y=(int *)malloc(vsize);
int i,j,k=0,e=0;
 int r=0;
int rp[5],r_ptr[5];
 for(i=0;i<M;i++)
 {
     r_ptr[i]=k;
     rp[i]=r_ptr[i];
   printf("row is %d k is %d  rptr is %d ",i,k, rp[i]);
     for(j=0;j<4;j++)
     {
     if(A[i][j]!=0)
     {
         csr[k++]=A[i][j];
         col_index[e++]=j;
     }
        
     }
 
 }

printf("\n");
r_ptr[4]=k;
 //printf("\n hi e is %d, k is %d, r is %d \n",e,k,r);
for(int t=0;t<5;t++)
{
    printf("%d ",r_ptr[t]);
}

  int *d_A,*d_X,*d_Y,*rptr,*colindex;
X[0]=10;X[1]=20;X[2]=30;X[3]=40;
Y[0]=2;Y[1]=4;Y[2]=5;Y[3]=6;
 i=5;
 hipMalloc((void **) &d_A, sizeof(int)*(k));
    hipMemcpy(d_A, csr, sizeof(int)*(k), hipMemcpyHostToDevice);
hipMalloc((void **) &d_X, vsize);
    hipMemcpy(d_X, X, vsize, hipMemcpyHostToDevice);
hipMalloc((void **) &d_Y, vsize);
    hipMemcpy(d_Y, Y, vsize, hipMemcpyHostToDevice);
  hipMalloc((void **) &colindex, sizeof(int)*e);
    hipMemcpy(colindex, col_index, sizeof(int)*e, hipMemcpyHostToDevice);
  hipMalloc((void **) &rptr, (sizeof(int)*i));
    hipMemcpy(rptr, r_ptr, (sizeof(int)*i), hipMemcpyHostToDevice);

kcsr<<<1,M>>> (d_A, d_X, d_Y, colindex, rptr,M);

 hipMemcpy(Y, d_Y, vsize, hipMemcpyDeviceToHost);
  for(i=0;i<M;i++)
  printf("\n%d ",Y[i]);
 return 0;
 }