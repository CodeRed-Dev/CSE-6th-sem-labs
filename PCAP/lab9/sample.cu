/*
#include <stdio.h>

#define N 2

__global__ void matrixAdd(int (*a)[N], int (*b)[N], int (*c)[N])
{
    int col = blockIdx.x;
    for (int row = 0; row < N; row++)
    {
        c[row][col] = a[row][col] + b[row][col];
    }
}

int main()
{
    int a[N][N] = {{1, 2}, {3, 4}};
    int b[N][N] = {{5, 6}, {7, 8}};
    int c[N][N];

    int (*dev_a)[N], (*dev_b)[N], (*dev_c)[N];

    // Allocate memory on the GPU
    cudaMalloc((void**)&dev_a, N*N*sizeof(int));
    cudaMalloc((void**)&dev_b, N*N*sizeof(int));
    cudaMalloc((void**)&dev_c, N*N*sizeof(int));

    // Copy matrices a and b from CPU to GPU
    cudaMemcpy(dev_a, a, N*N*sizeof(int), cudaMemcpyHostToDevice);
    cudaMemcpy(dev_b, b, N*N*sizeof(int), cudaMemcpyHostToDevice);

    // Launch kernel to add matrices
    dim3 grid(N, 1, 1);
    dim3 block(1, 1, 1);
    matrixAdd<<<grid, block>>>(dev_a, dev_b, dev_c);

    // Copy result matrix c from GPU to CPU
    cudaMemcpy(c, dev_c, N*N*sizeof(int), cudaMemcpyDeviceToHost);

    // Print result matrix c
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    // Free memory on the GPU
    cudaFree(dev_a);
    cudaFree(dev_b);
    cudaFree(dev_c);

    return 0;
}
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

__global__ void add_matrices_row(float *a, float *b, float *c, int num_rows, int num_cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num_rows) {
        for (int j = 0; j < num_cols; j++) {
            c[tid*num_cols+j] = a[tid*num_cols+j] + b[tid*num_cols+j];
        }
    }
}

int main() {
    int num_rows, num_cols, size;
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Get matrix dimensions from user
    printf("Enter number of rows: ");
    scanf("%d", &num_rows);
    printf("Enter number of columns: ");
    scanf("%d", &num_cols);

    // Allocate memory for matrices on host
    size = num_rows * num_cols * sizeof(float);
    h_a = (float*) malloc(size);
    h_b = (float*) malloc(size);
    h_c = (float*) malloc(size);

    // Get matrix elements from user
    printf("Enter elements of matrix A:\n");
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            scanf("%f", &h_a[i*num_cols+j]);
        }
    }

    printf("Enter elements of matrix B:\n");
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            scanf("%f", &h_b[i*num_cols+j]);
        }
    }

    // Allocate memory for matrices on device
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    // Copy matrices from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Add matrices using one thread per row
    add_matrices_row<<<(num_rows+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c, num_rows, num_cols);

    // Copy result matrix from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Result matrix using one thread per row:\n");
    for (int i = 0; i < num_rows; i++){
        for (int j = 0; j < num_cols; j++) {
            printf("%.2f ", h_c[i*num_cols+j]);
        }
        printf("\n");
    }

    // Free memory on host and device
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
