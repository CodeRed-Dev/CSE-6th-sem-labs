#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 4 //matrix size

__global__ void matrix_mult(float *a, float *b, float *c) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < N; ++i) {
        float sum = 0.0f;
        for (int j = 0; j < N; ++j) {
            sum += a[row * N + j] * b[j * N + i];
        }
        c[row * N + i] = sum;
    }
}

int main() {
    int h_a[N * N] = {1, 2, 3, 4};
    int h_b[N * N] = {5, 6, 7, 8};
    int h_c[N * N];
    float *d_a, *d_b, *d_c;

/*
    // Initialize matrices
    for (int i = 0; i < N * N; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }
*/
    // Allocate memory on the GPU
    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * N * sizeof(float));
    hipMalloc(&d_c, N * N * sizeof(float));

    // Copy matrices from CPU to GPU
    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with appropriate block and thread dimensions
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(N, 1, 1);
    matrix_mult<<<gridDim, blockDim>>>(d_a, d_b, d_c);

    // Copy result from GPU to CPU
    hipMemcpy(h_c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < N * N; ++i) {
        printf("%f ", h_c[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }

    // Free memory on GPU and CPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
