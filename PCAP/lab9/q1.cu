/*
#include<cuda.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 2

__global__ void matrixAdd(int *a, int *b, int *c, int n)
{
    int row = blockIdx.x;
    for (int col = 0; col < n; col++)
    {
        int index = row * n + col;
        c[index] = a[index] + b[index];
    }
}

int main()
{
    int a[N*N] = {1, 2, 3, 4};
    int b[N*N] = {5, 6, 7, 8};
    int c[N*N];

    int *dev_a, *dev_b, *dev_c;

    // Allocate memory on the GPU
    cudaMalloc((void**)&dev_a, N*N*sizeof(int));
    cudaMalloc((void**)&dev_b, N*N*sizeof(int));
    cudaMalloc((void**)&dev_c, N*N*sizeof(int));

    // Copy matrices a and b from CPU to GPU
    cudaMemcpy(dev_a, a, N*N*sizeof(int), cudaMemcpyHostToDevice);
    cudaMemcpy(dev_b, b, N*N*sizeof(int), cudaMemcpyHostToDevice);

    // Launch kernel to add matrices
    dim3 grid(N, 1, 1);
    dim3 block(1, 1, 1);
    matrixAdd<<<grid, block>>>(dev_a, dev_b, dev_c, N);

    // Copy result matrix c from GPU to CPU
    cudaMemcpy(c, dev_c, N*N*sizeof(int), cudaMemcpyDeviceToHost);

    // Print result matrix c
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", c[i*N + j]);
        }
        printf("\n");
    }

    // Free memory on the GPU
    cudaFree(dev_a);
    cudaFree(dev_b);
    cudaFree(dev_c);

    return 0;
}
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32
#define N 2

__global__ void add_matrices_row(float *a, float *b, float *c, int num_rows, int num_cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num_rows) {
        for (int j = 0; j < num_cols; j++) {
            c[tid*num_cols+j] = a[tid*num_cols+j] + b[tid*num_cols+j];
        }
    }
}

int main() {
    int num_rows, num_cols, size;
    //float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    float a[N*N] = {1, 2, 3, 4};
    float b[N*N] = {5, 6, 7, 8};
    float c[N*N];

    // Get matrix dimensions from user
   /* printf("Enter number of rows: ");
    scanf("%d", &num_rows);
    printf("Enter number of columns: ");
    scanf("%d", &num_cols);
*/



    // Get matrix elements from user
    /*printf("Enter elements of matrix A:\n");
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            scanf("%f", &h_a[i*num_cols+j]);
        }
    }

    printf("Enter elements of matrix B:\n");
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            scanf("%f", &h_b[i*num_cols+j]);
        }
    }
*/


    // Allocate memory for matrices on device
    size = N*N*sizeof(float);
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);
    hipMalloc((void**)&a, size);
    hipMalloc((void**)&b, size);
    hipMalloc((void**)&c, size);

    // Copy matrices from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Add matrices using one thread per row
    add_matrices_row<<<(num_rows+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_b, d_c, N, N);

    // Copy result matrix from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Result matrix using one thread per row:\n");
    for (int i = 0; i < num_rows; i++){
        for (int j = 0; j < num_cols; j++) {
            printf("%.2f ", c[i*num_cols+j]);
        }
        printf("\n");
    }

    // Free memory on host and device
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
