#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 2

__global__ void matrixAddElement(int *a, int *b, int *c, int num_rows, int num_cols)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num_rows * num_cols)
    {
        int i = tid / num_cols;
        int j = tid % num_cols;
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    int a[N * N] = {1, 2, 3, 4};
    int b[N * N] = {5, 6, 7, 8};
    int c[N * N];

    int *dev_a, *dev_b, *dev_c;

    // Allocate memory on the GPU
    hipMalloc((void **)&dev_a, N * N * sizeof(int));
    hipMalloc((void **)&dev_b, N * N * sizeof(int));
    hipMalloc((void **)&dev_c, N * N * sizeof(int));

    // Copy matrices a and b from CPU to GPU
    hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to add matrices
    dim3 grid(1 + ((N * N - 1) / 256), 1, 1);
    dim3 block(256, 1, 1);
    matrixAddElement<<<grid, block>>>(dev_a, dev_b, dev_c, N, N);

    // Copy result matrix c from GPU to CPU
    hipMemcpy(c, dev_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result matrix c

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", c[i * N + j]);
        }
        printf("\n");
    }

    // Free memory on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
