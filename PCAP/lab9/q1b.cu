
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 2

__global__ void matrixAddCol(int *a, int *b, int *c, int n)
{
    int row = blockIdx.x;
    for (int col = 0; col < n; col++)
    {
        int index = row * n + col;
        c[index] = a[index] + b[index];
    }
}

int main()
{
    int a[N*N] = {1, 2, 3, 4};
    int b[N*N] = {5, 6, 7, 8};
    int c[N*N];

    int *dev_a, *dev_b, *dev_c;

    // Allocate memory on the GPU
    hipMalloc((void**)&dev_a, N*N*sizeof(int));
    hipMalloc((void**)&dev_b, N*N*sizeof(int));
    hipMalloc((void**)&dev_c, N*N*sizeof(int));

    // Copy matrices a and b from CPU to GPU
    hipMemcpy(dev_a, a, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*N*sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to add matrices
    dim3 grid(N, 1, 1);
    dim3 block(1, 1, 1);
    matrixAddCol<<<grid, block>>>(dev_a, dev_b, dev_c, N);

    // Copy result matrix c from GPU to CPU
    hipMemcpy(c, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);

    // Print result matrix c
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", c[i*N + j]);
        }
        printf("\n");
    }

    // Free memory on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

