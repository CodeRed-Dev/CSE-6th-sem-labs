
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 2

__global__ void add_matrices_row(float *a, float *b, float *c, int num_rows, int num_cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num_rows) {
        for (int j = 0; j < num_cols; j++) {
            c[tid*num_cols+j] = a[tid*num_cols+j] + b[tid*num_cols+j];
        }
    }
}

int main()
{
    float a[N * N] = {1.0, 2.0, 3.0, 4.0};
    float b[N * N] = {5.0, 6.0, 7.0, 8.0};
    float c[N * N];

    float *dev_a, *dev_b, *dev_c;

    // Allocate memory on the GPU
    hipMalloc((void **)&dev_a, N * N * sizeof(float));
    hipMalloc((void **)&dev_b, N * N * sizeof(float));
    hipMalloc((void **)&dev_c, N * N * sizeof(float));

    // Copy matrices a and b from CPU to GPU
    hipMemcpy(dev_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to add matrices
    dim3 grid(1 + ((N - 1) / 256), 1, 1);
    dim3 block(256, 1, 1);
    add_matrices_row<<<grid, block>>>(dev_a, dev_b, dev_c, N, N);

    // Copy result matrix c from GPU to CPU
    hipMemcpy(c, dev_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result matrix c
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%f ", c[i * N + j]);
        }
        printf("\n");
    }

    // Free memory on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
