#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void add(int *a, int *b, int *c,int *n)
{
    for(int i=0;i<*n;i++){
        c[i]=a[i]+b[i];
    }
}
int main(void)
{
  // host copies of variables a, b & c
    int n;
    printf("Enter size: ");
    scanf("%d",&n);
    int c[n]={0};
    int a[n],b[n];
    printf("Enter array1:\n");
    for(int i=0;i<n;i++){
        scanf("%d",&a[i]);
    }
    printf("Enter array2:\n");
    for(int i=0;i<n;i++){
        scanf("%d",&b[i]);
    }
    int *d_a, *d_b, *d_c,*d_n; // device copies of variables a, b & c
    int size = sizeof(int)*n;
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_n, sizeof(int));
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    
    // Copy inputs to device
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    int x=ceil(float(n)/float(256));
    printf("%d\n",x);
    add<<<x, 256>>>(d_a, d_b, d_c,d_n);
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++){
        printf("%d ",c[i]);
    }
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}