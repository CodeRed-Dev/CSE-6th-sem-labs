#include<stdio.h>
#include"hip/hip_runtime.h"


//using BLOCK of size const size 10

const short N = 10;

__global__ void vecadd(int *d_a, int *d_b,int *d_c)
{
	unsigned short tid = blockIdx.x;
	
	if(tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
	}
}

int main (void)
{
	int Host_a[N], Host_b[N], Host_c[N];

	int *d_a , *d_b, *d_c ;

      
      hipMalloc((void **)&d_a , N*sizeof(int));
      hipMalloc((void **)&d_b , N*sizeof(int));
      hipMalloc((void **)&d_c , N*sizeof(int));

      //random values in the host array on cpu
      for ( int i = 0; i <N ; i++ )
      {
            Host_a[i] = i ;
            Host_b[i] = 2*i ; 
      }

      
      hipMemcpy (d_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice);
      hipMemcpy (d_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice);
      //just switch places of 1 and N for the execution of N threads
      //curently for block size as N
      vecadd<<<N,1>>> (d_a, d_b , d_c ) ;
      hipMemcpy(Host_c , d_c , N*sizeof(int) , hipMemcpyDeviceToHost);

      for ( int i = 0; i<N; i++ )
                  printf ("%d + %d = %d\n", Host_a[i] , Host_b[i] , Host_c[i] ) ;

      hipFree (d_a) ;
      hipFree (d_b) ;
      hipFree (d_c) ;
      return 0 ;

}
