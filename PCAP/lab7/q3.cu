

#include <stdio.h>

#include "hip/hip_runtime.h"



__host__ __device__ int strlen1(char *A)

{

    int i, c = 0;

    for (i = 0; A[i] != '\0'; i = i + 1)

    {

        c = c + 1;
    }

    return c;
}

__global__ void vecConv(float *A, float *B, float *C, int n, int l1, int kl, int p)

{

    int i = (threadIdx.x + blockDim.x * blockIdx.x);

    float sum = 0;

    int j = 0, c, k, l;

    c = kl / 2;

    if (i >= p && i < n - p)

    {

        sum = sum + A[i] * B[c];

        for (l = 0, j = c - 1, k = i - 1; l < c; j--, k--, l++)

            sum = sum + A[k] * B[j];

        for (l = 0, j = c + 1, k = i + 1; l < c && j < kl; j++, k++, l++)

            sum = sum + A[k] * B[j];
    }

    C[i - p] = sum;
}

void vecAdd(float *A, float *B, float *C, int n, int l1, int kl, int p)

{
    int size = n * sizeof(float);

    int size1 = kl * sizeof(float);

    int size2 = l1 * sizeof(float);

    float *d_A;

    float *d_B, *d_C;

    hipMalloc((void **)&d_A, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_B, size1);

    hipMemcpy(d_B, B, size1, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_C, size2);

    // Kernel invocation code

    vecConv<<<1, 32>>>(d_A, d_B, d_C, n, l1, kl, p);

    hipMemcpy(C, d_C, size2, hipMemcpyDeviceToHost);

    // Free device memory for A, B, C

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()

{

    // Host data buffers

    int p, d = 1, s = 1;

    int l1, kl, i;

    l1 = 7;

    kl = 3;

    p = (s * (l1 - 1) - l1 + d * (kl - 1) + 1) / 2;

    printf(" p is %d", p);

    float *A = NULL; // Input array

    float *B = NULL; // Input array

    float *C = NULL; // Output array

    // Elements in each array

    const int elements = l1 + 2 * p; // size of host data buffers

    // Compute the size of the data in bytes

    printf(" ele is %d", elements);

    size_t datasize = sizeof(float) * elements;

    size_t datasize1 = sizeof(float) * kl;

    size_t datasize2 = sizeof(float) * l1;

    // Dynamically allocate space for input/output host data buffers

    A = (float *)malloc(datasize);

    B = (float *)malloc(datasize1);

    C = (float *)malloc(datasize2);

    // Initialize the input data

    for (i = 0; i < p; i++)

    {

        A[i] = 0;

        A[elements - i - 1] = 0;
    }

    int k = p;

    A[k] = 10;
    k = k + 1;

    A[k] = 50;
    k = k + 1;

    A[k] = 60;
    k = k + 1;

    A[k] = 10;
    k = k + 1;

    A[k] = 20;
    k = k + 1;

    A[k] = 40;
    k = k + 1;

    A[k] = 30;
    k = k + 1;

    // A[k]=10;k=k+1;

    // A[k]=5;k=k+1;

    // A[k]=9;k=k+1;

    // A[k]=8;k=k+1;

    // A[k]=4;k=k+1;

    // A[k]=3;k=k+1;

    for (i = 0; i < elements; i++)

    {

        printf("%f ", A[i]);
    }

    printf("\n");

    B[0] = 0.333;
    B[1] = 0.333;
    B[2] = 0.333;
    B[3] = 0.333; // B[4]=5;

    for (i = 0; i < kl; i++)

    {

        printf("%f ", B[i]);
    }

    vecAdd(A, B, C, elements, l1, kl, p);

    // Verify the output

    bool result = true;

    printf("\n array is ");

    for (int i = 0; i < l1; i++)

    {

        printf(" %f ", C[i]);
    }

    free(A);
    free(B);
    free(C);

    return 0;
}