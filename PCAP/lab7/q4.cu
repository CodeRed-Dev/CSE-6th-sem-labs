#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<math.h>
__global__ void sinval(float a[], float c[],int *n)
{
    for(int i=0;i<*n;i++){
        c[i]=sin(a[i]);
    }
}
int main(void)
{
  // host copies of variables a, b & c
    int n;
    printf("Enter size: ");
    scanf("%d",&n);
    float c[n]={0};
    float a[n];
    printf("Enter array1:\n");
    for(int i=0;i<n;i++){
        scanf("%f",&a[i]);
    }
    float *d_a, *d_c;
    int *d_n; // device copies of variables a, b & c
    int size = sizeof(float)*n;
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_n, sizeof(int));
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    
    // Copy inputs to device
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    sinval<<<1, n>>>(d_a,d_c,d_n);
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++){
        printf("%f ",c[i]);
    }
    // Cleanup
    hipFree(d_a);
    hipFree(d_n);
    hipFree(d_c);
    return 0;
}